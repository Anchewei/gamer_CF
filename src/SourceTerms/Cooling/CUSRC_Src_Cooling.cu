#include "CUFLU.h"



// external functions and GPU-related set-up
#ifdef __HIPCC__

#include "CUAPI.h"
#include "CUFLU_Shared_FluUtility.cu"
#include "CUDA_ConstMemory.h"

#endif // #ifdef __HIPCC__


// local function prototypes
#ifndef __HIPCC__

void Src_SetAuxArray_Cooling( double [], int [] );
void Src_SetConstMemory_Cooling( const double AuxArray_Flt[], const int AuxArray_Int[],
                                       double *&DevPtr_Flt, int *&DevPtr_Int );
void Src_SetFunc_Cooling( SrcFunc_t & );
void Src_WorkBeforeMajorFunc_Cooling( const int lv, const double TimeNew, const double TimeOld, const double dt,
                                            double AuxArray_Flt[], int AuxArray_Int[] );
void Src_End_Cooling();

#endif



/********************************************************
1. Template of a user-defined source term
   --> Enabled by the runtime option "SRC_USER"

2. This file is shared by both CPU and GPU

   CUSRC_Src_Cooling.cu -> CPU_Src_Cooling.cpp

3. Four steps are required to implement a source term

   I.   Set auxiliary arrays
   II.  Implement the source-term function
   III. [Optional] Add the work to be done every time
        before calling the major source-term function
   IV.  Set initialization functions

4. The source-term function must be thread-safe and
   not use any global variable
********************************************************/



// =======================
// I. Set auxiliary arrays
// =======================

//-------------------------------------------------------------------------------------------------------
// Function    :  Src_SetAuxArray_Cooling
// Description :  Set the auxiliary arrays AuxArray_Flt/Int[]
//
// Note        :  1. Invoked by Src_Init_Cooling()
//                2. AuxArray_Flt/Int[] have the size of SRC_NAUX_USER defined in Macro.h (default = 10)
//                3. Add "#ifndef __HIPCC__" since this routine is only useful on CPU
//
// Parameter   :  AuxArray_Flt/Int : Floating-point/Integer arrays to be filled up
//
// Return      :  AuxArray_Flt/Int[]
//-------------------------------------------------------------------------------------------------------
#ifndef __HIPCC__
void Src_SetAuxArray_Cooling( double AuxArray_Flt[], int AuxArray_Int[] )
{

   
   AuxArray_Flt[0] = ISM_UNIT_T2;
   AuxArray_Flt[1] = UNIT_T;
   AuxArray_Flt[2] = 1e-2;
   AuxArray_Flt[3] = 1e50;
   AuxArray_Flt[4] = GAMMA - 1.0;
   AuxArray_Flt[5] = 1 / AuxArray_Flt[4];
   AuxArray_Flt[6] = Const_kB;
   AuxArray_Flt[7] = POW( 10.0, 0.1 );
   AuxArray_Flt[8] = 0.2;

} // FUNCTION : Src_SetAuxArray_Cooling
#endif // #ifndef __HIPCC__



// ======================================
// II. Implement the source-term function
// ======================================

//-------------------------------------------------------------------------------------------------------
// Function    :  Src_Cooling
// Description :  Major source-term function
//
// Note        :  1. Invoked by CPU/GPU_SrcSolver_IterateAllCells()
//                2. See Src_SetAuxArray_Cooling() for the values stored in AuxArray_Flt/Int[]
//                3. Shared by both CPU and GPU
//
// Parameter   :  fluid             : Fluid array storing both the input and updated values
//                                    --> Including both active and passive variables
//                B                 : Cell-centered magnetic field
//                SrcTerms          : Structure storing all source-term variables
//                dt                : Time interval to advance solution
//                dh                : Grid size
//                x/y/z             : Target physical coordinates
//                TimeNew           : Target physical time to reach
//                TimeOld           : Physical time before update
//                                    --> This function updates physical time from TimeOld to TimeNew
//                MinDens/Pres/Eint : Density, pressure, and internal energy floors
//                EoS               : EoS object
//                AuxArray_*        : Auxiliary arrays (see the Note above)
//
// Return      :  fluid[]
//-----------------------------------------------------------------------------------------
GPU_DEVICE_NOINLINE
static void Src_Cooling( real fluid[], const real B[],
                               const SrcTerms_t *SrcTerms, const real dt, const real dh,
                               const double x, const double y, const double z,
                               const double TimeNew, const double TimeOld,
                               const real MinDens, const real MinPres, const real MinEint,
                               const EoS_t *EoS, const double AuxArray_Flt[], const int AuxArray_Int[] )
{

// check
#  ifdef GAMER_DEBUG
   if ( AuxArray_Flt == NULL )   printf( "ERROR : AuxArray_Flt == NULL in %s !!\n", __FUNCTION__ );
   if ( AuxArray_Int == NULL )   printf( "ERROR : AuxArray_Int == NULL in %s !!\n", __FUNCTION__ );
#  endif

   const real UNIT_T2          = AuxArray_Flt[0];
   const real Time2CGS         = AuxArray_Flt[1];
   const real Tmin             = AuxArray_Flt[2];
   const real Tmax             = AuxArray_Flt[3];
   const real Gamma_m1         = AuxArray_Flt[4];
   const real _Gamma_m1        = AuxArray_Flt[5];
   const real kB               = AuxArray_Flt[6];
   const real vardt            = AuxArray_Flt[7];
   const real varrel           = AuxArray_Flt[8];
   const bool CheckMinEint_Yes = true;
   const int Mode = 0;
   real Alpha_CT;
   real Etot, Emag, Eint_Code, Delta_Eint_Code;
   real Dens_Code, Dens_CGS;
   real Temp_CGS, Temp_Old_CGS;
   real Temp_Init_CGS, Temp_Update_CGS, Delta_Temp_CGS;
   real Temp_Code, Delta_Temp_Code;
   real CoolingRate0, CoolingRate1, dCoolingRatedTemp;
   real Delta_Time, Max_Delta_Time;
   real dt_CGS, Time = 0.0;
   real In_Flt[2] = {0};
   real Out[1] = {0};
   real Epsilon = 1e-5;
   int iterations = 0;

#  ifdef MHD
   Emag  = (real)0.5*( SQR(B[MAGX]) + SQR(B[MAGY]) + SQR(B[MAGZ]) );
#  else
   Emag  = (real)0.0;
#  endif
   
   Dens_Code = fluid[DENS];
   Eint_Code  = Hydro_Con2Eint( Dens_Code, fluid[MOMX], fluid[MOMY], fluid[MOMZ], fluid[ENGY], CheckMinEint_Yes, MinEint, Emag );
   Temp_Code  = Gamma_m1 * Eint_Code / Dens_Code;

   if ( Temp_Code <= 0.0 )
   {
      Temp_Init_CGS = Temp_Code * 1.4 * UNIT_T2;
      Temp_Update_CGS = 50.0;
      Delta_Temp_CGS = Temp_Update_CGS - Temp_Init_CGS;
      Delta_Temp_Code = Delta_Temp_CGS / UNIT_T2;
      Delta_Eint_Code = Delta_Temp_Code * Dens_Code * _Gamma_m1;
      fluid[ENGY] = Hydro_ConEint2Etot( Dens_Code, fluid[MOMX], fluid[MOMY], fluid[MOMZ], Eint_Code + Delta_Eint_Code, Emag );
      return;
   }

   Temp_CGS   = Temp_Code * UNIT_T2;
   Temp_CGS  = MIN( MAX( Temp_CGS, Tmin), Tmax );
   Temp_Init_CGS= Temp_CGS;
   dt_CGS = dt * Time2CGS;
   
   if ( Dens_CGS <= 1e-10 )   Dens_CGS = 1e-10;
   Alpha_CT = Dens_CGS * kB * _Gamma_m1;

   while( Time < dt_CGS )
   {
      if ( Temp_CGS < 0.0 )
      {
         Temp_CGS = MIN( 4000 / MAX(Dens_CGS, 1e-10), 8000);
      }
      Temp_Old_CGS = Temp_CGS;
      In_Flt[0] = Dens_CGS;
      In_Flt[1] = Temp_CGS;
      EoS->General_FuncPtr(Mode, Out, In_Flt, NULL, EoS->AuxArrayDevPtr_Flt, EoS->AuxArrayDevPtr_Int, EoS->Table);
      CoolingRate0 = Out[0];

      In_Flt[0] = Dens_CGS;
      In_Flt[1] = Temp_CGS * Epsilon;
      EoS->General_FuncPtr(Mode, Out, In_Flt, NULL, EoS->AuxArrayDevPtr_Flt, EoS->AuxArrayDevPtr_Int, EoS->Table);
      CoolingRate1 = Out[0];

      dCoolingRatedTemp = (CoolingRate1 - CoolingRate0) / (Temp_CGS * Epsilon);
      
      if (iterations == 0)
      {
         if ( dCoolingRatedTemp != 0.0 )
         {
            Delta_Time = FABS( 0.1 * Alpha_CT / dCoolingRatedTemp );
         }
         else
         {
            Delta_Time = 0.1 * dt_CGS;
         }

         Max_Delta_Time = dt_CGS - Time;
         
         if ( Delta_Time > 0.7 * Max_Delta_Time )
         {
            Delta_Time = Max_Delta_Time * ( 1.0 + 1e-12 );
         }
      }

      Delta_Temp_CGS = CoolingRate0 / ( Alpha_CT / Delta_Time - dCoolingRatedTemp );
      Epsilon = FABS(Delta_Temp_CGS / Temp_CGS);
      if ( Epsilon > 0.2 )
      {
         Delta_Temp_CGS = 0.2 * Temp_Code * Delta_Temp_CGS / FABS(Delta_Temp_CGS);
      }

      iterations ++;
      Temp_CGS += Delta_Temp_CGS;
      Time += Delta_Time;

      Delta_Time = vardt * varrel * Delta_Time / MAX( vardt * Epsilon, varrel );
      Max_Delta_Time = dt_CGS - Time;   
      if ( Delta_Time > 0.7 * Max_Delta_Time )
      {
         Delta_Time = Max_Delta_Time * ( 1.0 + 1e-12 );
      }

      if ( Temp_CGS < 0.0 )
      {
         Temp_CGS = 100.0;
      }
   }
   Temp_Update_CGS = Temp_CGS;
   
   Delta_Temp_CGS = Temp_Update_CGS - Temp_Init_CGS;
   Delta_Temp_Code = Delta_Temp_CGS / UNIT_T2;
   Delta_Eint_Code = Delta_Temp_Code * Dens_Code * _Gamma_m1;
   fluid[ENGY] = Hydro_ConEint2Etot( Dens_Code, fluid[MOMX], fluid[MOMY], fluid[MOMZ], Eint_Code + Delta_Eint_Code, Emag );

} // FUNCTION : Src_Cooling



// ==================================================
// III. [Optional] Add the work to be done every time
//      before calling the major source-term function
// ==================================================

//-------------------------------------------------------------------------------------------------------
// Function    :  Src_WorkBeforeMajorFunc_Cooling
// Description :  Specify work to be done every time before calling the major source-term function
//
// Note        :  1. Invoked by Src_WorkBeforeMajorFunc()
//                   --> By linking to "Src_WorkBeforeMajorFunc_User_Ptr" in Src_Init_Cooling()
//                2. Add "#ifndef __HIPCC__" since this routine is only useful on CPU
//
// Parameter   :  lv               : Target refinement level
//                TimeNew          : Target physical time to reach
//                TimeOld          : Physical time before update
//                                   --> The major source-term function will update the system from TimeOld to TimeNew
//                dt               : Time interval to advance solution
//                                   --> Physical coordinates : TimeNew - TimeOld == dt
//                                       Comoving coordinates : TimeNew - TimeOld == delta(scale factor) != dt
//                AuxArray_Flt/Int : Auxiliary arrays
//                                   --> Can be used and/or modified here
//                                   --> Must call Src_SetConstMemory_Cooling() after modification
//
// Return      :  AuxArray_Flt/Int[]
//-------------------------------------------------------------------------------------------------------
#ifndef __HIPCC__
void Src_WorkBeforeMajorFunc_Cooling( const int lv, const double TimeNew, const double TimeOld, const double dt,
                                            double AuxArray_Flt[], int AuxArray_Int[] )
{

// uncomment the following lines if the auxiliary arrays have been modified
//#  ifdef GPU
//   Src_SetConstMemory_Cooling( AuxArray_Flt, AuxArray_Int,
//                                     SrcTerms.User_AuxArrayDevPtr_Flt, SrcTerms.User_AuxArrayDevPtr_Int );
//#  endif

} // FUNCTION : Src_WorkBeforeMajorFunc_Cooling
#endif



// ================================
// IV. Set initialization functions
// ================================

#ifdef __HIPCC__
#  define FUNC_SPACE __device__ static
#else
#  define FUNC_SPACE            static
#endif

FUNC_SPACE SrcFunc_t SrcFunc_Ptr = Src_Cooling;

//-----------------------------------------------------------------------------------------
// Function    :  Src_SetFunc_Cooling
// Description :  Return the function pointer of the CPU/GPU source-term function
//
// Note        :  1. Invoked by Src_Init_Cooling()
//                2. Call-by-reference
//                3. Use either CPU or GPU but not both of them
//
// Parameter   :  SrcFunc_CPU/GPUPtr : CPU/GPU function pointer to be set
//
// Return      :  SrcFunc_CPU/GPUPtr
//-----------------------------------------------------------------------------------------
#ifdef __HIPCC__
__host__
void Src_SetFunc_Cooling( SrcFunc_t &SrcFunc_GPUPtr )
{
   CUDA_CHECK_ERROR(  hipMemcpyFromSymbol( &SrcFunc_GPUPtr, HIP_SYMBOL(SrcFunc_Ptr), sizeof(SrcFunc_t) )  );
}

#elif ( !defined GPU )

void Src_SetFunc_Cooling( SrcFunc_t &SrcFunc_CPUPtr )
{
   SrcFunc_CPUPtr = SrcFunc_Ptr;
}

#endif // #ifdef __HIPCC__ ... elif ...



#ifdef __HIPCC__
//-------------------------------------------------------------------------------------------------------
// Function    :  Src_SetConstMemory_Cooling
// Description :  Set the constant memory variables on GPU
//
// Note        :  1. Adopt the suggested approach for CUDA version >= 5.0
//                2. Invoked by Src_Init_Cooling() and, if necessary, Src_WorkBeforeMajorFunc_Cooling()
//                3. SRC_NAUX_USER is defined in Macro.h
//
// Parameter   :  AuxArray_Flt/Int : Auxiliary arrays to be copied to the constant memory
//                DevPtr_Flt/Int   : Pointers to store the addresses of constant memory arrays
//
// Return      :  c_Src_User_AuxArray_Flt[], c_Src_User_AuxArray_Int[], DevPtr_Flt, DevPtr_Int
//---------------------------------------------------------------------------------------------------
void Src_SetConstMemory_Cooling( const double AuxArray_Flt[], const int AuxArray_Int[],
                                       double *&DevPtr_Flt, int *&DevPtr_Int )
{

// copy data to constant memory
   CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_Src_Cooling_AuxArray_Flt), AuxArray_Flt, SRC_NAUX_COOLING*sizeof(double) )  );
   CUDA_CHECK_ERROR(  hipMemcpyToSymbol(HIP_SYMBOL( c_Src_Cooling_AuxArray_Int), AuxArray_Int, SRC_NAUX_COOLING*sizeof(int   ) )  );

// obtain the constant-memory pointers
   CUDA_CHECK_ERROR(  hipGetSymbolAddress( (void **)&DevPtr_Flt, c_Src_Cooling_AuxArray_Flt) );
   CUDA_CHECK_ERROR(  hipGetSymbolAddress( (void **)&DevPtr_Int, c_Src_Cooling_AuxArray_Int) );

} // FUNCTION : Src_SetConstMemory_Cooling
#endif // #ifdef __HIPCC__



#ifndef __HIPCC__

//-----------------------------------------------------------------------------------------
// Function    :  Src_Init_Cooling
// Description :  Initialize a user-specified source term
//
// Note        :  1. Set auxiliary arrays by invoking Src_SetAuxArray_*()
//                   --> Copy to the GPU constant memory and store the associated addresses
//                2. Set the source-term function by invoking Src_SetFunc_*()
//                   --> Unlike other modules (e.g., EoS), here we use either CPU or GPU but not
//                       both of them
//                3. Set the function pointers "Src_WorkBeforeMajorFunc_User_Ptr" and "Src_End_User_Ptr"
//                4. Invoked by Src_Init()
//                   --> Enable it by linking to the function pointer "Src_Init_User_Ptr"
//                5. Add "#ifndef __HIPCC__" since this routine is only useful on CPU
//
// Parameter   :  None
//
// Return      :  None
//-----------------------------------------------------------------------------------------
void Src_Init_Cooling()
{
   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ...\n", __FUNCTION__ );
// set the auxiliary arrays
   Src_SetAuxArray_Cooling( Src_Cooling_AuxArray_Flt, Src_Cooling_AuxArray_Int );

// copy the auxiliary arrays to the GPU constant memory and store the associated addresses
#  ifdef GPU
   Src_SetConstMemory_Cooling( Src_Cooling_AuxArray_Flt, Src_Cooling_AuxArray_Int,
                               SrcTerms.Cooling_AuxArrayDevPtr_Flt, SrcTerms.Cooling_AuxArrayDevPtr_Int );
#  else
   SrcTerms.Cooling_AuxArrayDevPtr_Flt = Src_Cooling_AuxArray_Flt;
   SrcTerms.Cooling_AuxArrayDevPtr_Int = Src_Cooling_AuxArray_Int;
#  endif

// set the major source-term function
   Src_SetFunc_Cooling( SrcTerms.Cooling_FuncPtr );

   if ( MPI_Rank == 0 )    Aux_Message( stdout, "%s ... done\n", __FUNCTION__ );
} // FUNCTION : Src_Init_Cooling



//-----------------------------------------------------------------------------------------
// Function    :  Src_End_Cooling
// Description :  Free the resources used by a user-specified source term
//
// Note        :  1. Invoked by Src_End()
//                   --> Enable it by linking to the function pointer "Src_End_User_Ptr"
//                2. Add "#ifndef __HIPCC__" since this routine is only useful on CPU
//
// Parameter   :  None
//
// Return      :  None
//-----------------------------------------------------------------------------------------
void Src_End_Cooling()
{


} // FUNCTION : Src_End_Cooling

#endif // #ifndef __HIPCC__
